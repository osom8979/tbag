#include "hip/hip_runtime.h"
/**
 * @file   CudaRaw.cu
 * @brief  CUDA raw implementation.
 * @author zer0
 * @date   2017-08-01
 *
 * @warning
 *  Don't use the modern C++(0x, 11, ...);
 */

#include <libtbag/gpu/cuda/CudaRaw.h>
#include <cstdio>
#include <cassert>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifndef TB_CUDA_DEVICE
#define TB_CUDA_DEVICE  __device__
#endif

#ifndef TB_CUDA_HOST
#define TB_CUDA_HOST  __host__
#endif

#ifndef TB_CUDA_GLOBAL
#define TB_CUDA_GLOBAL  __global__
#endif

#ifndef TB_CUDA_INLINE
#define TB_CUDA_INLINE  __forceinline__
#endif

static bool isTbagCudaRawVerbose()
{
#if defined(TBAG_CUDA_RAW_VERBOSE)
    return true;
#else
    return false;
#endif
}

#ifndef tbCudaRawVerbose
#define tbCudaRawVerbose(...)     \
    if (isTbagCudaRawVerbose()) { \
        printf(__VA_ARGS__);      \
    } /* -- END -- */
#endif

// ---------------
namespace __impl {
// ---------------

template <typename T>
bool tbCudaGetMaxPotentialBlockSize(int * result_grid_size, int * result_block_size, T func, std::size_t array_count,
                                    std::size_t dynamic_shared_mem_size = 0, int block_size_limit = 0)
{
    int block_size    = 0; // The launch configurator returned block size.
    int min_grid_size = 0; // The minimum grid size needed to achieve the maximum occupancy for a full device launch.
    int round_up_grid = 0; // The actual grid size needed, based on input size.

    hipError_t code = ::hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, func,
                                                            dynamic_shared_mem_size, block_size_limit);
    if (code != hipSuccess) {
        return false;
    }

    round_up_grid = (array_count + block_size - 1) / block_size; // Round up according to array size.
    round_up_grid = (round_up_grid > min_grid_size ? round_up_grid : min_grid_size);

    if (result_grid_size != TB_NULL) {
        *result_grid_size = round_up_grid;
    }
    if (result_block_size != TB_NULL) {
        *result_block_size = block_size;
    }
    return true;
}

/** Calculate theoretical occupancy. */
template <typename T>
bool tbCudaGetMaxPotentialBlockSizeOfTheoretical(float * result_occupancy, T func, int block_size,
                                                 std::size_t dynamic_shared_mem_size = 0)
{
    hipError_t code;

    int max_active_blocks = 0; // Returned occupancy.
    code = ::hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks, func,
                                                           block_size, dynamic_shared_mem_size);
    if (code != hipSuccess) {
        return false;
    }

    int device = 0;
    hipDeviceProp_t props;

    code = hipGetDevice(&device);
    if (code != hipSuccess) {
        return false;
    }

    code = hipGetDeviceProperties(&props, device);
    if (code != hipSuccess) {
        return false;
    }

    float const OCCUPANCY = (max_active_blocks * block_size / props.warpSize)
                            / (float)(props.maxThreadsPerMultiProcessor / props.warpSize);
    if (result_occupancy != TB_NULL) {
        *result_occupancy = OCCUPANCY;
    }
    return true;
}

static TB_CUDA_INLINE TB_CUDA_DEVICE int __global_index_1g_1b__()
{
    return (blockIdx.x * blockDim.x) /*GLOBAL_BLOCK_OFFSET*/
           + threadIdx.x /*LOCAL_THREAD_OFFSET*/;
}

//static TB_CUDA_INLINE TB_CUDA_DEVICE int __global_index_1g_2b__()
//{
//    return (blockIdx.x * (blockDim.x * blockDim.y)) /*GLOBAL_BLOCK_OFFSET*/
//           + (threadIdx.y * blockDim.x) /*LOCAL_THREAD_Y_OFFSET*/
//           + threadIdx.x /*LOCAL_THREAD_X_OFFSET*/;
//}
//
//static TB_CUDA_INLINE TB_CUDA_DEVICE int __global_index_1g_3b__()
//{
//    return (blockIdx.x * (blockDim.x * blockDim.y * blockDim.z)) /*GLOBAL_BLOCK_OFFSET*/
//           + (threadIdx.z * (blockDim.y * blockDim.x)) /*LOCAL_THREAD_X_OFFSET*/
//           + (threadIdx.y * blockDim.x) /*LOCAL_THREAD_Y_OFFSET*/
//           + threadIdx.x /*LOCAL_THREAD_X_OFFSET*/;
//}

// ------------------
} // namespace __impl
// ------------------

using namespace __impl;

template <typename T>
TB_CUDA_GLOBAL void tbCudaFillKernel(T * out, T data, int size)
{
    int const index = __global_index_1g_1b__();
    if (index < size) {
        out[index] = data;
    }
}

template <typename T, typename StreamType>
bool tbCudaFill(T * out, T data, int count, StreamType stream)
{
    int grid_size(0), block_size(0);
    if (tbCudaGetMaxPotentialBlockSize(&grid_size, &block_size, tbCudaFillKernel<T>, count) == false) {
        return TB_FALSE;
    }
    assert(grid_size  > 0);
    assert(block_size > 0);

    tbCudaRawVerbose("tbCudaFill() GRID: %d, BLOCK: %d\n", grid_size, block_size);

    tbCudaFillKernel<T><<<grid_size, block_size, 0, stream>>>(out, data, count);

    if (isTbagCudaRawVerbose()) {
        float occupancy = 0;
        if (tbCudaGetMaxPotentialBlockSizeOfTheoretical(&occupancy, tbCudaFillKernel<T>, block_size)) {
            tbCudaRawVerbose("tbCudaFill() Launched blocks of size %d -> Theoretical occupancy: %f\n",
                             block_size, occupancy);
        }
    }
    return TB_TRUE;
}

// @formatter:off
tbBOOL tbCudaFill_i(int * out, int data, int count, void * stream)
{ return tbCudaFill(out, data, count, (hipStream_t)stream) ? TB_TRUE : TB_FALSE; }
tbBOOL tbCudaFill_u(unsigned * out, unsigned data, int count, void * stream)
{ return tbCudaFill(out, data, count, (hipStream_t)stream) ? TB_TRUE : TB_FALSE; }
tbBOOL tbCudaFill_f(float * out, float data, int count, void * stream)
{ return tbCudaFill(out, data, count, (hipStream_t)stream) ? TB_TRUE : TB_FALSE; }
tbBOOL tbCudaFill_d(double * out, double data, int count, void * stream)
{ return tbCudaFill(out, data, count, (hipStream_t)stream) ? TB_TRUE : TB_FALSE; }
// @formatter:on

template <typename T>
TB_CUDA_GLOBAL void tbCudaAddKernel(T const * in1, T const * in2, T * out, int size)
{
    int const index = __global_index_1g_1b__();
    if (index < size) {
        out[index] = in1[index] + in2[index];
    }
}

template <typename T, typename StreamType>
bool tbCudaAdd(T const * in1, T const * in2, T * out, int count, StreamType stream)
{
    int grid_size(0), block_size(0);
    if (tbCudaGetMaxPotentialBlockSize(&grid_size, &block_size, tbCudaAddKernel<T>, count) == false) {
        return TB_FALSE;
    }
    assert(grid_size  > 0);
    assert(block_size > 0);

    tbCudaAddKernel<T><<<grid_size, block_size, 0, stream>>>(in1, in2, out, count);
    return TB_TRUE;
}

// @formatter:off
tbBOOL tbCudaAdd_i(int const * in1, int const * in2, int * out, int count, void * stream)
{ return tbCudaAdd(in1, in2, out, count, (hipStream_t)stream) ? TB_TRUE : TB_FALSE; }
tbBOOL tbCudaAdd_u(unsigned const * in1, unsigned const * in2, unsigned * out, int count, void * stream)
{ return tbCudaAdd(in1, in2, out, count, (hipStream_t)stream) ? TB_TRUE : TB_FALSE; }
tbBOOL tbCudaAdd_f(float const * in1, float const * in2, float * out, int count, void * stream)
{ return tbCudaAdd(in1, in2, out, count, (hipStream_t)stream) ? TB_TRUE : TB_FALSE; }
tbBOOL tbCudaAdd_d(double const * in1, double const * in2, double * out, int count, void * stream)
{ return tbCudaAdd(in1, in2, out, count, (hipStream_t)stream) ? TB_TRUE : TB_FALSE; }
// @formatter:on

