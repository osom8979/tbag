#include "hip/hip_runtime.h"
/**
 * @file   CudaRaw.cu
 * @brief  CUDA raw implementation.
 * @author zer0
 * @date   2017-08-01
 */

#include <libtbag/gpu/cuda/CudaRaw.h>
#include <hip/hip_runtime.h>

template <typename T>
__global__ void tbCudaAddKernel(T const * v1, T const * v2, T * result, int size)
{
    int const index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < size) {
        result[index] = v1[index] + v2[index];
    }
}

int tbCudaAdd1f(float const * gpu_v1, float const * gpu_v2, float * gpu_result, int size)
{
    int block_size    = 0;
    int min_grid_size = 0;
    int grid_size     = 0;

    ::hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, kernel, 0, size);
    grid_size = (size + block_size - 1) / block_size;
    grid_size = (min_grid_size > grid_size ? min_grid_size : grid_size);
    tbCudaAddKernel<float><<<grid_size, block_size>>>(gpu_v1, gpu_v2, gpu_result, size);

    return TB_CUDA_TRUE;
}

int tbCudaAdd1d(double const * gpu_v1, double const * gpu_v2, double * gpu_result, int size)
{
    int block_size    = 0;
    int min_grid_size = 0;
    int grid_size     = 0;

    ::hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, kernel, 0, size);
    grid_size = (size + block_size - 1) / block_size;
    grid_size = (min_grid_size > grid_size ? min_grid_size : grid_size);
    tbCudaAddKernel<double><<<grid_size, block_size>>>(gpu_v1, gpu_v2, gpu_result, size);

    return TB_CUDA_TRUE;
}

