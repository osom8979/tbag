#include "hip/hip_runtime.h"
/**
 * @file   CudaRaw.cu
 * @brief  CUDA raw implementation.
 * @author zer0
 * @date   2017-08-01
 */

#include <libtbag/gpu/cuda/CudaRaw.h>
#include <hip/hip_runtime.h>

__global__ void add(int * a, int * b, int * c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

int tbCudaAddByGpu(int const * lh, int const * rh, int * result, int size)
{
    int * device_lh;
    int * device_rh;
    int * device_result;

    ::hipMalloc((void**)&device_lh, size);
    ::hipMalloc((void**)&device_rh, size);
    ::hipMalloc((void**)&device_result, size);

    ::hipMemcpy(device_lh, lh, size, ::hipMemcpyHostToDevice);
    ::hipMemcpy(device_rh, rh, size, ::hipMemcpyHostToDevice);

    add<<<size, 1>>>(device_lh, device_rh, device_result);

    ::hipMemcpy(result, device_rh, size, ::hipMemcpyDeviceToHost);

    ::hipFree(device_lh);
    ::hipFree(device_rh);
    ::hipFree(device_result);

    return TB_CUDA_TRUE;
}

