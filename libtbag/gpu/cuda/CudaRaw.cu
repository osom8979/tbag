#include "hip/hip_runtime.h"
/**
 * @file   CudaRaw.cu
 * @brief  CUDA raw implementation.
 * @author zer0
 * @date   2017-08-01
 */

#include <libtbag/gpu/cuda/CudaRaw.h>
#include <hip/hip_runtime.h>

__global__ void addByOnlyBlocks(int * lh, int * rh, int * result)
{
    result[blockIdx.x] = lh[blockIdx.x] + rh[blockIdx.x];
}

int tbCudaAddByGpu(int const * lh, int const * rh, int * result, int size)
{
    int const MEMORY_SIZE = size * sizeof(int);
    int * device_lh;
    int * device_rh;
    int * device_result;

    ::hipMalloc((void**)&device_lh    , MEMORY_SIZE);
    ::hipMalloc((void**)&device_rh    , MEMORY_SIZE);
    ::hipMalloc((void**)&device_result, MEMORY_SIZE);

    ::hipMemcpy(device_lh, lh, MEMORY_SIZE, ::hipMemcpyHostToDevice);
    ::hipMemcpy(device_rh, rh, MEMORY_SIZE, ::hipMemcpyHostToDevice);

    addByOnlyBlocks<<<size, 1>>>(device_lh, device_rh, device_result);
    ::hipDeviceSynchronize();

    ::hipMemcpy(result, device_result, MEMORY_SIZE, ::hipMemcpyDeviceToHost);

    ::hipFree(device_lh);
    ::hipFree(device_rh);
    ::hipFree(device_result);

    return TB_CUDA_TRUE;
}

