#include "hip/hip_runtime.h"
/**
 * @file   CudaRaw.cu
 * @brief  CUDA raw implementation.
 * @author zer0
 * @date   2017-08-01
 *
 * @warning
 *  Don't use the modern C++(0x, 11, ...);
 */

#include <libtbag/gpu/cuda/CudaRaw.h>
#include <cstdio>
#include <cassert>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifndef TB_CUDA_DEVICE
#define TB_CUDA_DEVICE  __device__
#endif

#ifndef TB_CUDA_GLOBAL
#define TB_CUDA_GLOBAL  __global__
#endif

#ifndef TB_CUDA_INLINE
#define TB_CUDA_INLINE  __forceinline__
#endif

static bool isTbagCudaRawVerbose()
{
#if defined(TBAG_CUDA_RAW_VERBOSE)
    return true;
#else
    return false;
#endif
}

#ifndef tCudaRawVerbose
#define tCudaRawVerbose(condition, ...) \
    if (condition) { \
        ::sprintf(stdout, __VA_ARGS__)\
    }
#endif

// ---------------
namespace __impl {
// ---------------

template <typename T>
bool tbCudaGetMaxPotentialBlockSize(int * result_grid_size, int * result_block_size, T func, std::size array_count,
                                    std::size_t dynamic_shared_mem_size = 0, int block_size_limit = 0)
{
    int block_size    = 0; // The launch configurator returned block size.
    int min_grid_size = 0; // The minimum grid size needed to achieve the maximum occupancy for a full device launch.
    int round_up_grid = 0; // The actual grid size needed, based on input size.

    hipError_t code = ::hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, func,
                                                            dynamic_shared_mem_size, block_size_limit);
    if (code != hipSuccess) {
        return false;
    }
    round_up_grid = (array_count + block_size - 1) / block_size; // Round up according to array size.
    round_up_grid = (round_up_grid > min_grid_size ? round_up_grid : min_grid_size);
    tCudaRawVerbose(isTbagCudaRawVerbose(), "tbCudaGetMaxPotentialBlockSize() GRID: %d, BLOCK: %d",
                    round_up_grid, block_size);
    if (result_grid_size != TB_NULL) {
        *result_grid_size = round_up_grid;
    }
    if (result_block_size != TB_NULL) {
        *result_block_size = block_size;
    }
    return true;
}

static TB_CUDA_INLINE TB_CUDA_DEVICE int __global_index_1g_1b__()
{
    return (blockIdx.x * blockDim.x) /*GLOBAL_BLOCK_OFFSET*/
           + threadIdx.x /*LOCAL_THREAD_OFFSET*/;
}

static TB_CUDA_INLINE TB_CUDA_DEVICE int __global_index_1g_2b__()
{
    return (blockIdx.x * (blockDim.x * blockDim.y)) /*GLOBAL_BLOCK_OFFSET*/
           + (threadIdx.y * blockDim.x) /*LOCAL_THREAD_Y_OFFSET*/
           + threadIdx.x /*LOCAL_THREAD_X_OFFSET*/;
}

static TB_CUDA_INLINE TB_CUDA_DEVICE int __global_index_1g_3b__()
{
    return (blockIdx.x * (blockDim.x * blockDim.y * blockDim.z)) /*GLOBAL_BLOCK_OFFSET*/
           + (threadIdx.z * (blockDim.y * blockDim.x)) /*LOCAL_THREAD_X_OFFSET*/
           + (threadIdx.y * blockDim.x) /*LOCAL_THREAD_Y_OFFSET*/
           + threadIdx.x /*LOCAL_THREAD_X_OFFSET*/;
}

template <typename T>
TB_CUDA_GLOBAL void tbCudaAddKernel(T const * v1, T const * v2, T * result, unsigned size)
{
    int const index = __global_index_1g_1b__();
    if (index < size) {
        result[index] = v1[index] + v2[index];
    }
}

template <typename T, typename StreamType>
tbBOOL tbCudaAdd(T const * v1, T const * v2, T * result, unsigned size, StreamType stream)
{
    int grid_size(0), block_size(0);
    if (tbCudaGetMaxPotentialBlockSize(&grid_size, &block_size, tbCudaAddKernel<T>) == false) {
        return TB_FALSE;
    }
    assert(grid_size > 0);
    assert(block_size > 0);

    tbCudaAddKernel<T><<<grid_size, block_size, 0, stream>>>(v1, v2, gpu_result, size);
    return TB_TRUE;
}

// ------------------
} // namespace __impl
// ------------------

tbBOOL tbCudaAdd1i(int const * gpu_v1, int const * gpu_v2, int * gpu_result, unsigned size, void * stream)
{
    return __impl::tbCudaAdd(gpu_v1, gpu_v2, gpu_result, size, (hipStream_t)stream);
}

tbBOOL tbCudaAdd1u(unsigned const * gpu_v1, unsigned const * gpu_v2, unsigned * gpu_result, unsigned size, void * stream)
{
    return __impl::tbCudaAdd(gpu_v1, gpu_v2, gpu_result, size, (hipStream_t)stream);
}

tbBOOL tbCudaAdd1f(float const * gpu_v1, float const * gpu_v2, float * gpu_result, unsigned size, void * stream)
{
    return __impl::tbCudaAdd(gpu_v1, gpu_v2, gpu_result, size, (hipStream_t)stream);
}

tbBOOL tbCudaAdd1d(double const * gpu_v1, double const * gpu_v2, double * gpu_result, unsigned size, void * stream)
{
    return __impl::tbCudaAdd(gpu_v1, gpu_v2, gpu_result, size, (hipStream_t)stream);
}

