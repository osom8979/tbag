#include "hip/hip_runtime.h"
/**
 * @file   CudaRaw.cu
 * @brief  CUDA raw implementation.
 * @author zer0
 * @date   2017-08-01
 *
 * @warning
 *  Don't use the modern C++(0x, 11, ...);
 */

#include <libtbag/gpu/cuda/CudaRaw.h>
#include <cstdio>
#include <cassert>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifndef TB_CUDA_DEVICE
#define TB_CUDA_DEVICE  __device__
#endif

#ifndef TB_CUDA_HOST
#define TB_CUDA_HOST  __host__
#endif

#ifndef TB_CUDA_GLOBAL
#define TB_CUDA_GLOBAL  __global__
#endif

#ifndef TB_CUDA_INLINE
#define TB_CUDA_INLINE  __forceinline__
#endif

static bool isTbagCudaRawVerbose()
{
#if defined(TBAG_CUDA_RAW_VERBOSE)
    return true;
#else
    return false;
#endif
}

#ifndef tbCudaRawVerbose
#define tbCudaRawVerbose(condition, ...) \
    if (condition) { printf(__VA_ARGS__); }
#endif

// ---------------
namespace __impl {
// ---------------

template <typename T>
bool tbCudaGetMaxPotentialBlockSize(int * result_grid_size, int * result_block_size, T func, std::size_t array_count,
                                    std::size_t dynamic_shared_mem_size = 0, int block_size_limit = 0)
{
    int block_size    = 0; // The launch configurator returned block size.
    int min_grid_size = 0; // The minimum grid size needed to achieve the maximum occupancy for a full device launch.
    int round_up_grid = 0; // The actual grid size needed, based on input size.

    hipError_t code = ::hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, func,
                                                            dynamic_shared_mem_size, block_size_limit);
    if (code != hipSuccess) {
        return false;
    }
    round_up_grid = (array_count + block_size - 1) / block_size; // Round up according to array size.
    round_up_grid = (round_up_grid > min_grid_size ? round_up_grid : min_grid_size);
    tbCudaRawVerbose(isTbagCudaRawVerbose(), "tbCudaGetMaxPotentialBlockSize() GRID: %d, BLOCK: %d", round_up_grid, block_size);
    if (result_grid_size != TB_NULL) {
        *result_grid_size = round_up_grid;
    }
    if (result_block_size != TB_NULL) {
        *result_block_size = block_size;
    }
    return true;
}

static TB_CUDA_INLINE TB_CUDA_DEVICE int __global_index_1g_1b__()
{
    return (blockIdx.x * blockDim.x) /*GLOBAL_BLOCK_OFFSET*/
           + threadIdx.x /*LOCAL_THREAD_OFFSET*/;
}

//static TB_CUDA_INLINE TB_CUDA_DEVICE int __global_index_1g_2b__()
//{
//    return (blockIdx.x * (blockDim.x * blockDim.y)) /*GLOBAL_BLOCK_OFFSET*/
//           + (threadIdx.y * blockDim.x) /*LOCAL_THREAD_Y_OFFSET*/
//           + threadIdx.x /*LOCAL_THREAD_X_OFFSET*/;
//}
//
//static TB_CUDA_INLINE TB_CUDA_DEVICE int __global_index_1g_3b__()
//{
//    return (blockIdx.x * (blockDim.x * blockDim.y * blockDim.z)) /*GLOBAL_BLOCK_OFFSET*/
//           + (threadIdx.z * (blockDim.y * blockDim.x)) /*LOCAL_THREAD_X_OFFSET*/
//           + (threadIdx.y * blockDim.x) /*LOCAL_THREAD_Y_OFFSET*/
//           + threadIdx.x /*LOCAL_THREAD_X_OFFSET*/;
//}

template <typename T>
TB_CUDA_GLOBAL void tbCudaAddKernel(T const * v1, T const * v2, T * r, int size)
{
    int const index = __global_index_1g_1b__();
    if (index < size) {
        r[index] = v1[index] + v2[index];
    }
}

template <typename T, typename StreamType>
bool tbCudaAdd(T const * v1, T const * v2, T * r, int count, StreamType stream)
{
    int grid_size(0), block_size(0);
    if (tbCudaGetMaxPotentialBlockSize(&grid_size, &block_size, tbCudaAddKernel<T>, count) == false) {
        return TB_FALSE;
    }
    assert(grid_size > 0);
    assert(block_size > 0);

    tbCudaAddKernel<T><<<grid_size, block_size, 0, stream>>>(v1, v2, r, count);
    return TB_TRUE;
}

// ------------------
} // namespace __impl
// ------------------

tbBOOL tbCudaAdd1i(int const * v1, int const * v2, int * r, int count, void * stream)
{ return __impl::tbCudaAdd(gpu_v1, v2, r, count, (hipStream_t)stream) ? TB_TRUE : TB_FALSE; }
tbBOOL tbCudaAdd1u(unsigned const * v1, unsigned const * v2, unsigned * r, int count, void * stream)
{ return __impl::tbCudaAdd(gpu_v1, v2, r, count, (hipStream_t)stream) ? TB_TRUE : TB_FALSE; }
tbBOOL tbCudaAdd1f(float const * v1, float const * v2, float * r, int count, void * stream)
{ return __impl::tbCudaAdd(gpu_v1, v2, r, count, (hipStream_t)stream) ? TB_TRUE : TB_FALSE; }
tbBOOL tbCudaAdd1d(double const * v1, double const * v2, double * r, int count, void * stream)
{ return __impl::tbCudaAdd(gpu_v1, v2, r, count, (hipStream_t)stream) ? TB_TRUE : TB_FALSE; }

