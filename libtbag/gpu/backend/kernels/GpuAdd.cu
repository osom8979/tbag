#include "hip/hip_runtime.h"
/**
 * @file   GpuAdd.cpp
 * @brief  GpuAdd class implementation.
 * @author zer0
 * @date   2018-01-07
 */

#include <libtbag/gpu/backend/kernels/GpuAdd.hpp>
#include <libtbag/log/Log.hpp>

#include <hip/hip_runtime.h>

// -------------------
NAMESPACE_LIBTBAG_OPEN
// -------------------

namespace gpu     {
namespace backend {
namespace kernels {

void addByCuda(float const * v1, float const * v2, float * result, std::size_t count)
{
}

void addByCuda(double const * v1, double const * v2, double * result, std::size_t count)
{
}

} // namespace kernels
} // namespace backend
} // namespace gpu

// --------------------
NAMESPACE_LIBTBAG_CLOSE
// --------------------

